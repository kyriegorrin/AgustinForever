#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//Afegim un tamany per defecte
//Imprescindible que sigui potencia de 2
#define NUM_THREADS 1024
#define NUM_BLOCKS 32768
#define N NUM_THREADS*NUM_BLOCKS

//Macro per a swap
#define SWAP(_i, _ixj){\
	int aux = vector[_i];\
	vector[_i] = vector[_ixj];\
	vector[_ixj] = aux;}

//Kernel per a bitonic sort
__global__ void bitonicSortKernel(int *vector, int j, int k){
	int i, ixj;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	ixj = i ^ j;

	if((ixj) > i){
		if((i & k) == 0 && vector[i] > vector[ixj]){
			SWAP(i, ixj);
			//int aux = vector[i];
			//vector[i] = vector[ixj];
			//vector[ixj] = aux;
		}
		if((i & k) != 0 && vector[i] < vector[ixj]){
			SWAP(i, ixj);
			//int aux = vector[i];
			//vector[i] = vector[ixj];
			//vector[ixj] = aux;
		}
	}
}

//Funcio iterativa de bitonic sort
void bitonicSort(int *vector){
	int j, k;

	dim3 numBlocks(NUM_BLOCKS, 1);
	dim3 numThreads(NUM_THREADS, 1);

	for(k = 2; k <= N; k = 2*k){
		//Els shifts son equivalents de dividir entre 2
		for(j = k >> 1; j > 0; j = j >> 1){ 
			bitonicSortKernel<<<numBlocks, numThreads>>>(vector, j, k);
		}
	}
}

int testOrdenacio(int *vector){
	int ordenat = 1;
	int i;
	for(i = 0; i < N -1 && ordenat; ++i){
		if(vector[i] > vector[i+1]) ordenat = 0;
	}
	return ordenat;
}

int main(int argc, char **argv) {
	//Generacio dels parametres del vector
	int n = N;
	//if(argc > 1) n = atoi(argv[1]); 
	int *vector, *vectorDevice, *vectorAux;

	hipEvent_t E0, E1, E2, E3;

	hipEventCreate(&E0);
	hipEventCreate(&E1);
	hipEventCreate(&E2);
	hipEventCreate(&E3);
	
	unsigned int NUM_BYTES = N*sizeof(int);

	//Reserva de memoria per als vectors
	hipHostMalloc(&vector, NUM_BYTES, hipHostMallocDefault);
	hipHostMalloc(&vectorAux, NUM_BYTES, hipHostMallocDefault);
	
	//Inicialitzacio amb valors random
	int i;
	srand(time(NULL));
	for(i = 0; i < N; ++i){
		vector[i] = rand();
	}

	hipEventRecord(E0, 0);
	hipEventSynchronize(E0);
	
	hipMalloc((int **)&vectorDevice, NUM_BYTES);

	hipEventRecord(E1, 0);
	hipEventSynchronize(E1);

	//Pas del vector de host a device
	hipMemcpy(vectorDevice, vector, NUM_BYTES, hipMemcpyHostToDevice);

	//Fem sort del vector
	bitonicSort(vector);

	hipEventRecord(E2, 0);
	hipEventSynchronize(E2);

	//Pas del vector de device a host
	hipMemcpy(vector, vectorDevice, NUM_BYTES, hipMemcpyDeviceToHost);

	//Test per veure si la ordenacio es correcte
	if(testOrdenacio(vector)) printf("Agustin is happy\n");
	else printf("Agustin te deniega el curso PUMPS\n");

	//Alliberacio de memoria
	hipFree(vector);
	hipFree(vectorDevice);

	hipDeviceSynchronize();
	hipEventRecord(E3, 0);
	hipEventSynchronize(E3);

	//Timing
	float tempsTotal;
	hipEventElapsedTime(&tempsTotal, E0, E3);
	
	printf("Temps: %f", tempsTotal);

	//Destrueix events
	hipEventDestroy(E0);
	hipEventDestroy(E1);
	hipEventDestroy(E2);
	hipEventDestroy(E3);

}


