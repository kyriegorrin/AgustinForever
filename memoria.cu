
#include <hip/hip_runtime.h>
#include <stdio.h>
int main() {
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("%d\n", prop.sharedMemPerBlock);	

	}



}
