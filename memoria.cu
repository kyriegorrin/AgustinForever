
#include <hip/hip_runtime.h>
#include <stdio.h>
int main() {
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("%lu\n", prop.sharedMemPerBlock);	

	}
}
